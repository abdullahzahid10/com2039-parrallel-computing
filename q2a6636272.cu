/*************************************
**Question 2a
*************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>

void helloCPU(){
    printf("Hello from the CPU.\n");
}

//I have used __global__ before void so the code is run of the GPU. In the method, I have printed a message to indicate the method is run from GPU.

__global__ void helloGPU()
{
  printf("Hello from the GPU.\n");
}

// Now to run the code on the GPU
int main()
{


//	An execution configuration with the helloGPU<<<>>> syntax will run this as a kernel from the GPU. The first iteration of 1 will indicate the amount of blocks in and the size of the blocks
	helloGPU<<<1, 1>>>();

//	The purpose of this is so that it will hinder the CPU Stream until the GPU kernels have finished.
	hipDeviceSynchronize();

	helloCPU();

	return 0;

}
