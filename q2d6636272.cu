/*************************************
**Question 2D
*************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

//Here i will refactor the the loop so that is a CUDA Kernel. This should only do 1 iteration of the original loop

__global__ void loop(int N)
{

//	The aim of idx and str is to ensure each iteration of numbers are printed out on one occasion/

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int str = gridDim.x * blockDim.x;

  for (int i = idx; i < N; i += str)
  {
    printf("This is iteration number %d\n", i);
  }
}

int main()
{
//The question requirement requires me to use 2 blocks of threads.

//	Amount of times the for loop will run till.
  int N = 10;

//  Here i used 2 blocks of thread followed by 1 which is the block size.
  loop<<<2, 1>>>(N);

//  Device sync method
  hipDeviceSynchronize();

  return 0;
}
