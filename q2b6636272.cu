/*************************************
**Question 2b
*************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>

// Function which will print our desired output.
__global__ void printSuccessForCorrectExecutionConfiguration()
{

  if(threadIdx.x == 1023 && blockIdx.x == 255)
  {
    printf("Success!\n");
  }

}

//Main method responsible for ptinting out whether everything has gone to plan
int main()
{

//	Here there are two numbers. 256 defines the amount of blocks and the amount of threads is set to 1024. We need to print success, providing the set conditions
//	are matche.
  printSuccessForCorrectExecutionConfiguration<<<256, 1024>>>();

//  Device sync method
  hipDeviceSynchronize();

  return 0;
}
