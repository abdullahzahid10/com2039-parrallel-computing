// In this question I will provide a complete implementation of a parallel algorithm that will Reduce a 1D array of
//elements into a single summary value, where the size of the input array will require the use
//of multiple blocks of threads. The reduction should give the sum of the list.


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

//Inclusion of Cuda Error Handling
inline hipError_t checkCuda(hipError_t proceed)
{
  if (proceed != hipSuccess) {
    fprintf(stderr, "Error... %s\n", hipGetErrorString(proceed));
    assert(proceed == hipSuccess);
  }
  return proceed;
}

// Initialising the method.
void initialisingStage(float number, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = number;
  }
}

//Add vectors into the array
__global__ void additionPhase(float *proceed, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    proceed[i] = a[i] + b[i];
  }
}

//Check elements are within the array to reduce
void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("1D Array has been successfully reduces.\n");
}

//Main method
int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

//  Pointers initialised
  float *a;
  float *b;
  float *c;

//  Error handling in the main method
  checkCuda( hipMallocManaged(&a,size));
  checkCuda( hipMallocManaged(&b,size));
  checkCuda( hipMallocManaged(&c,size));


//Initializing phase
  initialisingStage(3, a, N);
  initialisingStage(4, b, N);
  initialisingStage(0, c, N);

//  Initializing threads per block and the amount of blocks.
  size_t threadsPerBlock;
  size_t numberOfBlocks;

//  Putting per block as 256
  threadsPerBlock = 256;
  numberOfBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;

//  Add vectors into the array
  additionPhase<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);

//  Check for any errors and make synchronisations where required.
  checkCuda( hipGetLastError() );
  checkCuda( hipDeviceSynchronize() );

//  Sync all entities including threads.
  hipDeviceSynchronize();

//  Check everything is in order
  checkElementsAre(7, c, N);

//  Free up any memory which is allocated.
  hipFree(a);
  hipFree(b);
  hipFree(c);
}