/*************************************
**Question 2c
*************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Refactor `loop` to be a CUDA Kernel. The new kernel should
 * only do the work of 1 iteration of the original loop.
 */

//I have used __global__ before void so the code is run of the GPU. In the method the for loop is run from GPU.
__global__ void loop(int N)
{
  for (int i = 0; i < N; ++i)
  {
    printf("This is iteration number %d\n", i);
  }
}

int main()
{

//The execution content to set out amount of loops which will occur.

  int N = 10;

//  In the loop<<<>>> syntax, i put 1,1. The first one will be 1 block and the latter being 1 thread.
  loop<<<1, 1>>>(N);

  hipDeviceSynchronize();
  return 0;


}
