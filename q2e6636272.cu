/*************************************
**Question 2e
*************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>

// Initialise method with a for loop. Will print all values from 0 all the way to what integer is stored in N.
void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

// The purpose of this function is to double a providing that thread is lower than a.

__global__ void doubleElements(int *a, int N)
{
  int i;
  i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N)
  {
    a[i] *= 2;
  }
}

// A boolean method, which is pivotal for the code to meet its requirement. The code in here will check whether the a has doubled.
// Also notice that it has 2 return clauses, whereby, it will return true if the above clause is met, otherwise it will return the latter.
bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
//Initialising n to be 100 and initialising a pointer for a.
  int N = 100;
  int *a;

//  This var will store the amount of elements in the list.
  size_t size = N * sizeof(int);

//
  hipMallocManaged(&a, size);

//  a = (int *)malloc(size);

// Calling the function to initialize variables.
  init(a, N);

//  Initializing two key variables which will be used later to be 10.
  size_t threads_per_block = 10;
  size_t number_of_blocks = 10;

  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  hipDeviceSynchronize();

//  Boolean variable which will indicate as to the integers have been doubled.
  bool areDoubled = checkElementsAreDoubled(a, N);

  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  hipFree(a);
}
